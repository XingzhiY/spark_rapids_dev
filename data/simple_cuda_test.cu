
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// 一个简单的 CUDA 核函数，用于产生一些 GPU 活动
__global__ void simple_kernel() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    float val = 0.0f;
    // 执行一些任意的计算来让 GPU 保持繁忙
    for (int i = 0; i < 2000; ++i) {
        val += sinf(static_cast<float>(i)) * cosf(static_cast<float>(idx));
    }
    // 结果没有被存储，因为这只是为了分析活动
    printf("Hello World from GPU %d\n", idx);
}

int main() {
    // 定义网格和块维度
    // 启动足够多的线程以确保 GPU 被利用
    int blockSize = 8;
    int gridSize = 1;

    std::cout << ">>> 正在为 nsys 分析启动简单的 CUDA 核函数..." << std::endl;

    // 启动核函数
    simple_kernel<<<gridSize, blockSize>>>();

    // cudaDeviceSynchronize() 等待核函数执行完成。
    // 这对于确保应用程序在核函数结束前不退出非常重要。
    hipDeviceSynchronize();
    printf("Hello World from CPU\n");

    // 检查核函数执行期间是否有任何错误
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA 错误: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    std::cout << ">>> CUDA 核函数成功执行完毕。" << std::endl;

    return 0;
} 
